#include <cstring>
#include <iostream>
#include <memory>
#include <string>
#include <ctime>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>

__global__ void findNearestNeighborCosine(float *points, float *queries, float *max_cosine, int n, int num_queries, int dimensions) {
    extern __shared__ char shared[];
    float *s_cosine = (float*)shared;
    int *s_index = (int*)(shared + blockDim.x * sizeof(float));

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int qid = blockIdx.y;

    if (tid < n && qid < num_queries) {
        float dot_product = 0, query_magnitude = 0, point_magnitude = 0;
        for (int d = 0; d < dimensions; ++d) {
            int idx = tid * dimensions + d;
            int q_idx = qid * dimensions + d;
            dot_product += queries[q_idx] * points[idx];
            query_magnitude += queries[q_idx] * queries[q_idx];
            point_magnitude += points[idx] * points[idx];
        }
        query_magnitude = sqrt(query_magnitude);
        point_magnitude = sqrt(point_magnitude);

        float cosine_similarity = 0;
        if (query_magnitude > 0 && point_magnitude > 0) {
            cosine_similarity = dot_product / (query_magnitude * point_magnitude);
        }

        s_cosine[threadIdx.x] = cosine_similarity;
        s_index[threadIdx.x] = tid;
        __syncthreads();
    }
}

std::vector<std::vector<float>> read_matrix(FILE* fin, int row, int col) {
    std::vector<std::vector<float>> ret;
    for (int i = 0; i < row; ++i) {
        std::vector<float> curr;
        float tmp = 0;
        for (int j = 0; j < col; ++j) {
            fscanf(fin, "%f", &tmp);
            curr.push_back(tmp);
        }
        ret.push_back(curr);
    }
    return ret;
}

int main(int argc, char* argv[]) {
    FILE* fin = fopen(argv[1], "r");

    int n = 0, d = 0, m = 0;
    fscanf(fin, "%d%d%d", &d, &n, &m);

    double total_cosine_GPU_time = 0.0;

    clock_t start_time, end_time;

    std::vector<std::vector<float>> base = read_matrix(fin, n, d);
    std::vector<std::vector<float>> query = read_matrix(fin, m, d);

    float* flat_base = new float[n * d];
    float* flat_query = new float[m * d];
    for (int i = 0; i < n; ++i)
        memcpy(flat_base + i * d, base[i].data(), d * sizeof(float));
    for (int i = 0; i < m; ++i)
        memcpy(flat_query + i * d, query[i].data(), d * sizeof(float));

    
    float* d_base, * d_query, *d_max_cosine;

    start_time = clock();

    hipMalloc(&d_base, n * d * sizeof(float));
    hipMalloc(&d_query, m * d * sizeof(float));
    hipMalloc(&d_max_cosine, m * sizeof(float));

  
    float *max_cosine_host = new float[m];
    for (int i = 0; i < m; i++) {
        max_cosine_host[i] = -1.0f;
    }

    
    hipMemcpy(d_base, flat_base, n * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_query, flat_query, m * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_max_cosine, max_cosine_host, m * sizeof(float), hipMemcpyHostToDevice);

    
    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((n + threadsPerBlock.x - 1) / threadsPerBlock.x, m);

    
    int sharedMemSize = threadsPerBlock.x * (sizeof(float) + sizeof(int));
    findNearestNeighborCosine<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_base, d_query, d_max_cosine, n, m, d);

   
    hipMemcpy(max_cosine_host, d_max_cosine, m * sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(d_base);
    hipFree(d_query);
    hipFree(d_max_cosine);

    end_time = clock(); 

    total_cosine_GPU_time = static_cast<double>(end_time - start_time) / CLOCKS_PER_SEC;

    std::cout << "Total cosine similarity with GPU: " << total_cosine_GPU_time << std::endl;

    return 0;
}